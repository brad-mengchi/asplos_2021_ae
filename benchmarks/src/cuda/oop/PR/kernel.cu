#include "hip/hip_runtime.h"
__global__ void initContext(GraphChiContext* context, int vertices, int edges) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid == 0) {
	context->setNumIterations(0);
	context->setNumVertices(vertices);
	context->setNumEdges(edges);
    }
}

__global__ void initObject(ChiVertex<float, float> **vertex, GraphChiContext* context,
	int* row, int* col, int* inrow, int* incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
	int out_start = row[tid];
	int out_end;
	if (tid + 1 < context->getNumVertices()) {
	    out_end = row[tid + 1];
	} else {
	    out_end = context->getNumEdges();
	}
	int in_start = inrow[tid];
	int in_end;
	if (tid + 1 < context->getNumVertices()) {
	    in_end = inrow[tid + 1];
	} else {
	    in_end = context->getNumEdges();
	}
	int indegree = in_end - in_start;
	int outdegree = out_end - out_start;
	vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
	for (int i = in_start; i < in_end; i++) {
	    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
	}
	//for (int i = out_start; i < out_end; i++) {
	//    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
	//}
    }
}

__global__ void initOutEdge(ChiVertex<float, float> **vertex, GraphChiContext* context,
	int* row, int* col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
	int out_start = row[tid];
	int out_end;
	if (tid + 1 < context->getNumVertices()) {
	    out_end = row[tid + 1];
	} else {
	    out_end = context->getNumEdges();
	}
	//int in_start = inrow[tid];
	//int in_end;
	//if (tid + 1 < context->getNumVertices()) {
	//    in_end = inrow[tid + 1];
	//} else {
	//    in_end = context->getNumEdges();
	//}
	//int indegree = in_end - in_start;
	//int outdegree = out_end - out_start;
	//vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
	//for (int i = in_start; i < in_end; i++) {
	//    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
	//}
	for (int i = out_start; i < out_end; i++) {
	    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
	}
    }
}

__global__ void PageRank(ChiVertex<float, float> **vertex, GraphChiContext* context) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
	if (context->getNumIterations() == 0) {
	    vertex[tid]->setValue(1.0f);
	} else {
	    float sum = 0.0f;
	    for (int i = 0; i < vertex[tid]->numInEdges(); i++) {
		sum+= vertex[tid]->getInEdge(i)->getValue();
	    }
	    vertex[tid]->setValue(0.15f + 0.85f * sum);

	    /* Write my value (divided by my out-degree) to my out-edges so neighbors can read it. */
	    float outValue = vertex[tid]->getValue() / vertex[tid]->numOutEdges();
	    for(int i=0; i<vertex[tid]->numOutEdges(); i++) {
		vertex[tid]->getOutEdge(i)->setValue(outValue);
	    }
	}
	context->setNumIterations(context->getNumIterations() + 1);
    }
}

__global__ void copyBack(ChiVertex<float, float> **vertex, GraphChiContext* context,
	float *pagerank)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        pagerank[tid] = vertex[tid]->getValue();
    }
}
