#include "hip/hip_runtime.h"
#include "parse_oo.h"

void initContext(GraphChiContext *context, int vertices, int edges) {
    // int tid = blockDim.x * blockIdx.x + threadIdx.x;

    context->setNumIterations(0);
    context->setNumVertices(vertices);
    context->setNumEdges(edges);
}

void initObject(VirtVertex<int, int> *vertex, GraphChiContext *context,
                int *row, int *col, int *inrow, int *incol, obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        // vertex[tid].inEdgeDataArray =
        //     (ChiEdge<int> *)alloc->my_new<Edge<int>>(indegree);
        // vertex[tid].outEdgeDataArray =
        //     (ChiEdge<int> **)alloc->my_new<Edge<int> *>(outdegree);
        new (&vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree, alloc);

        vertex[tid].setValueV(INT_MAX);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid].setInEdgeV(i - in_start, incol[i], INT_MAX);
        }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}

void part0_initObject(VirtVertex<int, int> **vertex, GraphChiContext *context,
                      int *row, int *col, int *inrow, int *incol,
                      obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        vertex[tid] =
            (VirtVertex<int, int> *)alloc->my_new<ChiVertex<int, int>>();
    }
}
void part1_initObject(VirtVertex<int, int> **vertex, GraphChiContext *context,
                      int *row, int *col, int *inrow, int *incol,
                      obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   in_end = inrow[tid + 1];
        // } else {
        //   in_end = context->getNumEdges();
        // }
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid].inEdgeDataArray =
        //     (ChiEdge<myType> *)alloc->my_new<Edge<myType>>(indegree);
        // vertex[tid].outEdgeDataArray =
        //     (ChiEdge<myType> **)alloc->my_new<Edge<myType> *>(outdegree);
        // new (&vertex[tid]) ChiVertex<int, int>(tid, indegree,
        // outdegree,alloc);
        vertex[tid]->set_in_out(alloc);
        // vertex[tid].setValue(INT_MAX);
        // for (int i = in_start; i < in_end; i++) {
        //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
        // }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}
__global__ void part_kern0_initObject(VirtVertex<int, int> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;

        new (vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree);

        // vertex[tid].setValue(INT_MAX);
        // for (int i = in_start; i < in_end; i++) {
        //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
        // }
    }
}
__global__ void part_kern1_initObject(VirtVertex<int, int> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }

        vertex[tid]->setValueV(INT_MAX);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid]->setInEdgeV(i - in_start, incol[i], INT_MAX);
        }
    }
}
void initOutEdge(VirtVertex<int, int> **vertex, GraphChiContext *context,
                 int *row, int *col) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i],
                                    INT_MAX);
        }
    }
}

__global__ void kern_initObject(VirtVertex<int, int> *vertex,
                                GraphChiContext *context, int *row, int *col,
                                int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        new (&vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree);

        vertex[tid].setValueV(INT_MAX);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid].setInEdgeV(i - in_start, incol[i], INT_MAX);
        }

        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
    }
    //}
}
__global__ void kern_initOutEdge(VirtVertex<int, int> **vertex,
                                 GraphChiContext *context, int *row, int *col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //    in_end = inrow[tid + 1];
        //} else {
        //    in_end = context->getNumEdges();
        //}
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
        // for (int i = in_start; i < in_end; i++) {
        //    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
        //}

        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i],
                                    INT_MAX);
        }
    }
}

__managed__ __align__(16) char buf2[128];
template <class myType>
__global__ void vptrPatch(myType *array, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("-----\n");
    myType *obj;
    obj = new (buf2) myType();
    // void *p;
    // p=(void *)0x111111111;
    // memcpy(p, obj, sizeof(void *));
    // printf("---%p--\n", p);
    if (tid < n) {
        memcpy(&array[tid], obj, sizeof(void *));
        // printf("---%p--\n",p);
    }
}

__global__ void vptrPatch_Edge(ChiVertex<int, int> *vertex, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    Edge<int> *obj;
    obj = new (buf2) Edge<int>();

    if (tid < n)
        if (tid == 0)
            vertex[tid].vptrPatch(obj, 1);
        else
            vertex[tid].vptrPatch(obj, 1);
}
__managed__ range_tree_node *range_tree;
__managed__ unsigned tree_size_g;
__managed__ void *temp_copyBack;
__managed__ void *temp_Bfs;

__global__ void BFS(VirtVertex<int, int> **vertex, GraphChiContext *context,
                    int iteration) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    if (iteration == 0) {
      if (tid == 0) {

        switch (vertex[tid]->type) {
        case 0:
          vertex[tid]->setValueC(0);
          break;
        case 1:
          vertex[tid]->setValueV(0);
          break;
        }
        int numOutEdge;

        switch (vertex[tid]->type) {
        case 0:
          numOutEdge = vertex[tid]->numOutEdgesC();
          break;
        case 1:
          numOutEdge = vertex[tid]->numOutEdgesV();
          break;
        }
        for (int i = 0; i < numOutEdge; i++) {
          ChiEdge<int> *outEdge;

          switch (vertex[tid]->type) {
          case 0:
            outEdge = vertex[tid]->getOutEdgeC(i);
            break;
          case 1:
            outEdge = vertex[tid]->getOutEdgeV(i);
            break;
          }
          switch (outEdge->type) {
          case 0:
            outEdge->setValueC(1);
            break;
          case 1:
            outEdge->setValueV(1);
            break;
          }
        }
      }
    } else {
      int curmin;

      switch (vertex[tid]->type) {
      case 0:
        curmin = vertex[tid]->getValueC();
        break;
      case 1:
        curmin = vertex[tid]->getValueV();
        break;
      }
      int numInEdge;

      switch (vertex[tid]->type) {
      case 0:
        numInEdge = vertex[tid]->numInEdgesC();
        break;
      case 1:
        numInEdge = vertex[tid]->numInEdgesV();
        break;
      }
      for (int i = 0; i < numInEdge; i++) {
        ChiEdge<int> *inEdge;

        switch (vertex[tid]->type) {
        case 0:
          inEdge = vertex[tid]->getInEdgeC(i);
          break;
        case 1:
          inEdge = vertex[tid]->getInEdgeV(i);
          break;
        }
        switch (inEdge->type) {
        case 0:
          curmin = min(curmin, inEdge->getValueC());
          break;
        case 1:
          curmin = min(curmin, inEdge->getValueV());
          break;
        }
      }
      int vertValue;

      switch (vertex[tid]->type) {
      case 0:
        vertValue = vertex[tid]->getValueC();
        break;
      case 1:
        vertValue = vertex[tid]->getValueV();
        break;
      }
      if (curmin < vertValue) {

        switch (vertex[tid]->type) {
        case 0:
          vertex[tid]->setValueC(curmin);
          break;
        case 1:
          vertex[tid]->setValueV(curmin);
          break;
        }
        int numOutEdge;
        switch (vertex[tid]->type) {
        case 0:
          numOutEdge = vertex[tid]->numOutEdgesC();
          break;
        case 1:
          numOutEdge = vertex[tid]->numOutEdgesV();
          break;
        }

        for (int i = 0; i < numOutEdge; i++) {
          ChiEdge<int> *outEdge;

          switch (vertex[tid]->type) {
          case 0:
            outEdge = vertex[tid]->getOutEdgeC(i);
            break;
          case 1:
            outEdge = vertex[tid]->getOutEdgeV(i);
            break;
          }

          int edgeValue;

          switch (outEdge->type) {
          case 0:
            edgeValue = outEdge->getValueC();
            break;
          case 1:
            edgeValue = outEdge->getValueV();
            break;
          }
          if (edgeValue > curmin + 1) {
            switch (outEdge->type) {
            case 0:
              outEdge->setValueC(curmin + 1);
              break;
            case 1:
              outEdge->setValueV(curmin + 1);
              break;
            }
          }
        }
      }
    }
  }
}

__managed__ void *temp_vfun;
__global__ void vfunCheck(VirtVertex<int, int> *vertex) {
    void **vtable;
    unsigned tree_size = tree_size_g;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    vtable = get_vfunc(&vertex[tid], range_tree, tree_size);
    temp_vfun = vtable[1];
    vertex[tid].setIdV(155);
    temp_vfun = vtable[0];
    printf("%d\n", vertex[tid].getIdV());

    temp_vfun = vtable[3];
    vertex[tid].setValueV(999);
    temp_vfun = vtable[2];
    printf("%d\n", vertex[tid].getValueV());
    temp_vfun = vtable[4];
    printf("%d\n", vertex[tid].numInEdgesV());
    temp_vfun = vtable[5];
    printf("%d\n", vertex[tid].numOutEdgesV());
    temp_vfun = vtable[6];
    printf("%p\n", vertex[tid].getInEdgeV(0));
    temp_vfun = vtable[7];
    printf("%p\n", vertex[tid].getOutEdgeV(0));
}

void BFS_cpu(VirtVertex<int, int> *vertex, GraphChiContext *context) {
    int tid = 0;
    // printf("ffff\n");
    for (tid = 0; tid < context->getNumVertices(); tid++) {
        if (context->getNumIterations() == 0) {
            if (tid == 0) {
                vertex[tid].setValueV(0);
                int numOutEdge;
                numOutEdge = vertex[tid].numOutEdgesV();
                for (int i = 0; i < numOutEdge; i++) {
                    ChiEdge<int> *outEdge;
                    outEdge = vertex[tid].getOutEdgeV(i);
                    outEdge->setValueV(1);
                }
            }
        } else {
            int curmin;
            curmin = vertex[tid].getValueV();
            int numInEdge;
            numInEdge = vertex[tid].numInEdgesV();
            for (int i = 0; i < numInEdge; i++) {
                ChiEdge<int> *inEdge;
                inEdge = vertex[tid].getInEdgeV(i);
                curmin = min(curmin, inEdge->getValueV());
            }
            int vertValue;
            vertValue = vertex[tid].getValueV();
            if (curmin < vertValue) {
                vertex[tid].setValueV(curmin);
                int numOutEdge;
                numOutEdge = vertex[tid].numOutEdgesV();
                for (int i = 0; i < numOutEdge; i++) {
                    ChiEdge<int> *outEdge;
                    outEdge = vertex[tid].getOutEdgeV(i);
                    int edgeValue;
                    edgeValue = outEdge->getValueV();
                    if (edgeValue > curmin + 1) {
                        outEdge->setValueV(curmin + 1);
                    }
                }
            }
        }
        //  context->setNumIterations(context->getNumIterations() + 1);
    }
}

__global__ void copyBack(VirtVertex<int, int> **vertex,
                         GraphChiContext *context, int *index) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned tree_size = tree_size_g;
    // ChiVertex<int, int> *obj;
    // obj = new (buf2) ChiVertex<int, int>();
    // long ***mVtable = (long ***)&vertex[tid];
    // long ***mVtable2 = (long ***)obj;
    // //memcpy(&vertex[tid],obj,sizeof(void*));
    // printf("[%d]-obj %p vert %p\n",tid,*mVtable2,*mVtable);
    // *mVtable=*mVtable2;
    // printf("[%d]after obj %p vert %p\n",tid,*mVtable2,*mVtable);
    if (tid < context->getNumVertices()) {
        void **vtable = get_vfunc(vertex[tid], range_tree, tree_size);
        temp_copyBack = vtable[2];
        // printf("%d\n",index[tid]);
        index[tid] = vertex[tid]->getValueV();
        //  if(mVtable[0][0]!=mVtable2[0][0])
        //  printf("[%d]why !! obj %p vert
        //  %p\n",tid,mVtable[0][0],mVtable2[0][0]);
        // printf("%d\n",index[tid]);
    }
}
