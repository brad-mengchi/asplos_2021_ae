#include "hip/hip_runtime.h"
/*
__global__ void initContext(GraphChiContext* context, int vertices, int edges) {

        context->setNumIterations(0);
        context->setNumVertices(vertices);
        context->setNumEdges(edges);

}

__global__ void initObject(VirtVertex<int, int> **vertex, GraphChiContext*
context,
        int* row, int* col, int* inrow, int* incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        vertex[tid] = new ChiVertex<int, int>(tid, indegree, outdegree);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid]->setInEdge(i - in_start, incol[i], 0);
        }
        //for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}

__global__ void initOutEdge(VirtVertex<int, int> **vertex, GraphChiContext*
context,
        int* row, int* col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0);
        }
    }
}
*/
#include "parse_oo.h"

void initContext(GraphChiContext *context, int vertices, int edges) {
    // int tid = blockDim.x * blockIdx.x + threadIdx.x;

    context->setNumIterations(0);
    context->setNumVertices(vertices);
    context->setNumEdges(edges);
}

void initObject(VirtVertex<int, int> *vertex, GraphChiContext *context,
                int *row, int *col, int *inrow, int *incol, obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        // vertex[tid].inEdgeDataArray =
        //     (ChiEdge<int> *)alloc->my_new<Edge<int>>(indegree);
        // vertex[tid].outEdgeDataArray =
        //     (ChiEdge<int> **)alloc->my_new<Edge<int> *>(outdegree);
        new (&vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree, alloc);

        vertex[tid].setValueV(INT_MAX);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid].setInEdgeV(i - in_start, incol[i], INT_MAX);
        }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}

void part0_initObject(VirtVertex<int, int> **vertex, GraphChiContext *context,
                      int *row, int *col, int *inrow, int *incol,
                      obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        vertex[tid] =
            (VirtVertex<int, int> *)alloc->my_new<ChiVertex<int, int>>();
    }
}
void part1_initObject(VirtVertex<int, int> **vertex, GraphChiContext *context,
                      int *row, int *col, int *inrow, int *incol,
                      obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   in_end = inrow[tid + 1];
        // } else {
        //   in_end = context->getNumEdges();
        // }
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid].inEdgeDataArray =
        //     (ChiEdge<myType> *)alloc->my_new<Edge<myType>>(indegree);
        // vertex[tid].outEdgeDataArray =
        //     (ChiEdge<myType> **)alloc->my_new<Edge<myType> *>(outdegree);
        // new (&vertex[tid]) ChiVertex<int, int>(tid, indegree,
        // outdegree,alloc);
        vertex[tid]->set_in_out(alloc);
        // vertex[tid].setValue(INT_MAX);
        // for (int i = in_start; i < in_end; i++) {
        //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
        // }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}
__global__ void part_kern0_initObject(VirtVertex<int, int> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        new (vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree);

        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }

    // vertex[tid].setValue(INT_MAX);
    // for (int i = in_start; i < in_end; i++) {
    //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
    // }
}
__global__ void part_kern1_initObject(VirtVertex<int, int> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }

        for (int i = in_start; i < in_end; i++) {
            vertex[tid]->setInEdgeV(i - in_start, incol[i], 0);
        }
    }
    // for (int i = out_start; i < out_end; i++) {
    //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
    //}
}
void initOutEdge(VirtVertex<int, int> **vertex, GraphChiContext *context,
                 int *row, int *col) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i], 0);
        }
    }
}

__global__ void kern_initObject(VirtVertex<int, int> *vertex,
                                GraphChiContext *context, int *row, int *col,
                                int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        new (&vertex[tid]) ChiVertex<int, int>(tid, indegree, outdegree);

        vertex[tid].setValueV(INT_MAX);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid].setInEdgeV(i - in_start, incol[i], INT_MAX);
        }

        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
    }
    //}
}
__global__ void kern_initOutEdge(VirtVertex<int, int> **vertex,
                                 GraphChiContext *context, int *row, int *col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i], 0);
        }
    }
}

__managed__ range_tree_node *range_tree;
__managed__ unsigned tree_size_g;
__managed__ void *temp_copyBack;
__managed__ void *temp_CC;

__global__ void ConnectedComponent(VirtVertex<int, int> **vertex,
                                   GraphChiContext *context, int iteration) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    int numEdges;

    switch (vertex[tid]->type) {
    case 0:
      numEdges = vertex[tid]->numEdgesC();
      break;
    case 1:
      numEdges = vertex[tid]->numEdgesV();
      break;
    }
    if (iteration == 0) {
      int vid;

      switch (vertex[tid]->type) {
      case 0:
        vid = vertex[tid]->getIdC();
        break;
      case 1:
        vid = vertex[tid]->getIdV();
        break;
      }

      switch (vertex[tid]->type) {
      case 0:
        vertex[tid]->setValueC(vid);
        break;
      case 1:
        vertex[tid]->setValueV(vid);
        break;
      }
    }
    int curMin;

    switch (vertex[tid]->type) {
    case 0:
      curMin = vertex[tid]->getValueC();
      break;
    case 1:
      curMin = vertex[tid]->getValueV();
      break;
    }
    for (int i = 0; i < numEdges; i++) {
      ChiEdge<int> *edge;

      switch (vertex[tid]->type) {
      case 0:
        edge = vertex[tid]->edgeC(i);
        break;
      case 1:
        edge = vertex[tid]->edgeV(i);
        break;
      }
      int nbLabel;

      switch (edge->type) {
      case 0:
        nbLabel = edge->getValueC();
        break;
      case 1:
        nbLabel = edge->getValueV();
        break;
      }
      if (iteration == 0) {
        switch (edge->type) {
        case 0:
          nbLabel = edge->getVertexIdC(); // Note!
          break;
        case 1:
          nbLabel = edge->getVertexIdV(); // Note!
          break;
        }
      }
      if (nbLabel < curMin) {
        curMin = nbLabel;
      }
    }

    /**
     * Set my new label
     */

    switch (vertex[tid]->type) {
    case 0:
      vertex[tid]->setValueC(curMin);
      break;
    case 1:
      vertex[tid]->setValueV(curMin);
      break;
    }
    int label = curMin;

    /**
     * Broadcast my value to neighbors by writing the value to my edges.
     */
    if (iteration > 0) {
      for (int i = 0; i < numEdges; i++) {
        ChiEdge<int> *edge;

        switch (vertex[tid]->type) {
        case 0:
          edge = vertex[tid]->edgeC(i);
          break;
        case 1:
          edge = vertex[tid]->edgeV(i);
          break;
        }
        int edgeValue;

        switch (edge->type) {
        case 0:
          edgeValue = edge->getValueC();
          break;
        case 1:
          edgeValue = edge->getValueV();
          break;
        }
        if (edgeValue > label) {
          switch (edge->type) {
          case 0:
            edge->setValueC(label);
            break;
          case 1:
            edge->setValueV(label);
            break;
          }
        }
      }
    } else {
      // Special case for first iteration to avoid overwriting
      int numOutEdge;

      switch (vertex[tid]->type) {
      case 0:
        numOutEdge = vertex[tid]->numOutEdgesC();
        break;
      case 1:
        numOutEdge = vertex[tid]->numOutEdgesV();
        break;
      }
      for (int i = 0; i < numOutEdge; i++) {
        ChiEdge<int> *outEdge;

        switch (vertex[tid]->type) {
        case 0:
          outEdge = vertex[tid]->getOutEdgeC(i);
          break;
        case 1:
          outEdge = vertex[tid]->getOutEdgeV(i);
          break;
        }

        switch (outEdge->type) {
        case 0:
          outEdge->setValueC(label);
          break;
        case 1:
          outEdge->setValueV(label);
          break;
        }
      }
    }
  }
}

__global__ void copyBack(VirtVertex<int, int> **vertex,
                         GraphChiContext *context, int *cc) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned tree_size = tree_size_g;
    void **vtable;
    range_tree_node *table = range_tree;
    if (tid < context->getNumVertices()) {
        vtable = get_vfunc(vertex[tid], table, tree_size);
        temp_copyBack = vtable[1];
        cc[tid] = vertex[tid]->getValueV();
    }
}
