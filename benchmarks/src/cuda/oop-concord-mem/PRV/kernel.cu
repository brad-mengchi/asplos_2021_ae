#include "hip/hip_runtime.h"
#include "parse_oo.h"

void initContext(GraphChiContext *context, int vertices, int edges) {
    context->setNumIterations(0);
    context->setNumVertices(vertices);
    context->setNumEdges(edges);
}
void part0_initObject(VirtVertex<float, float> **vertex,
                      GraphChiContext *context, int *row, int *col, int *inrow,
                      int *incol, obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        vertex[tid] = (VirtVertex<float, float> *)
                          alloc->my_new<ChiVertex<float, float>>();
    }
}
void part1_initObject(VirtVertex<float, float> **vertex,
                      GraphChiContext *context, int *row, int *col, int *inrow,
                      int *incol, obj_alloc *alloc) {
    int tid = 0;

    for (tid = 0; tid < context->getNumVertices(); tid++) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   in_end = inrow[tid + 1];
        // } else {
        //   in_end = context->getNumEdges();
        // }
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid].inEdgeDataArray =
        //     (ChiEdge<myType> *)alloc->my_new<Edge<myType>>(indegree);
        // vertex[tid].outEdgeDataArray =
        //     (ChiEdge<myType> **)alloc->my_new<Edge<myType> *>(outdegree);
        // new (&vertex[tid]) ChiVertex<int, int>(tid, indegree,
        // outdegree,alloc);
        vertex[tid]->set_in_out(alloc);
        // vertex[tid].setValue(INT_MAX);
        // for (int i = in_start; i < in_end; i++) {
        //   vertex[tid].setInEdge(i - in_start, incol[i], INT_MAX);
        // }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}

__global__ void part_kern0_initObject(VirtVertex<float, float> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        new (vertex[tid]) ChiVertex<float, float>(tid, indegree, outdegree);
    }
}
__global__ void part_kern1_initObject(VirtVertex<float, float> **vertex,
                                      GraphChiContext *context, int *row,
                                      int *col, int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < context->getNumVertices()) {
        // int out_start = row[tid];
        // int out_end;
        // if (tid + 1 < context->getNumVertices()) {
        //   out_end = row[tid + 1];
        // } else {
        //   out_end = context->getNumEdges();
        // }

        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }

        for (int i = in_start; i < in_end; i++) {
            vertex[tid]->setInEdgeV(i - in_start, incol[i], 0.0f);
        }
    }
}
__global__ void kern_initOutEdge(VirtVertex<float, float> **vertex,
                                 GraphChiContext *context, int *row, int *col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //    in_end = inrow[tid + 1];
        //} else {
        //    in_end = context->getNumEdges();
        //}
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
        // for (int i = in_start; i < in_end; i++) {
        //    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
        //}

        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i], 0.0f);
        }
    }
}
__global__ void initObject(VirtVertex<float, float> **vertex,
                           GraphChiContext *context, int *row, int *col,
                           int *inrow, int *incol) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        int in_start = inrow[tid];
        int in_end;
        if (tid + 1 < context->getNumVertices()) {
            in_end = inrow[tid + 1];
        } else {
            in_end = context->getNumEdges();
        }
        int indegree = in_end - in_start;
        int outdegree = out_end - out_start;
        vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
        for (int i = in_start; i < in_end; i++) {
            vertex[tid]->setInEdgeV(i - in_start, incol[i], 0.0f);
        }
        // for (int i = out_start; i < out_end; i++) {
        //    vertex[tid]->setOutEdge(vertex, tid, i - out_start, col[i], 0.0f);
        //}
    }
}

__global__ void initOutEdge(VirtVertex<float, float> **vertex,
                            GraphChiContext *context, int *row, int *col) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        int out_start = row[tid];
        int out_end;
        if (tid + 1 < context->getNumVertices()) {
            out_end = row[tid + 1];
        } else {
            out_end = context->getNumEdges();
        }
        // int in_start = inrow[tid];
        // int in_end;
        // if (tid + 1 < context->getNumVertices()) {
        //    in_end = inrow[tid + 1];
        //} else {
        //    in_end = context->getNumEdges();
        //}
        // int indegree = in_end - in_start;
        // int outdegree = out_end - out_start;
        // vertex[tid] = new ChiVertex<float, float>(tid, indegree, outdegree);
        // for (int i = in_start; i < in_end; i++) {
        //    vertex[tid]->setInEdge(i - in_start, incol[i], 0.0f);
        //}
        for (int i = out_start; i < out_end; i++) {
            vertex[tid]->setOutEdgeV(vertex, tid, i - out_start, col[i], 0.0f);
        }
    }
}
__managed__ range_tree_node *range_tree;
__managed__ unsigned tree_size_g;
__managed__ void *temp_copyBack;
__managed__ void *temp_PR;

__global__ void PageRank(VirtVertex<float, float> **vertex,
                         GraphChiContext *context, int iteration) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < context->getNumVertices()) {
    if (iteration == 0) {

      switch (vertex[tid]->type) {
      case 0:
        vertex[tid]->setValueC(1.0f);
        break;
      case 1:
        vertex[tid]->setValueV(1.0f);
        break;
      }
    } else {
      float sum = 0.0f;
      int numInEdge;
	  
	  switch (vertex[tid]->type) {
		case 0:
		numInEdge = vertex[tid]->numInEdgesC();
		  break;
		case 1:
		numInEdge = vertex[tid]->numInEdgesV();
		  break;
		}
      for (int i = 0; i < numInEdge; i++) {
        ChiEdge<float> *inEdge;
		
		switch (vertex[tid]->type) {
			case 0:
			inEdge = vertex[tid]->getInEdgeC(i);
			  break;
			case 1:
			inEdge = vertex[tid]->getInEdgeV(i);
			  break;
			}
		
		
        switch (inEdge->type) {
			case 0:
			sum += inEdge->getValueC();
			  break;
			case 1:
			sum += inEdge->getValueV();
			  break;
			}
      }
      
	  switch (vertex[tid]->type) {
		case 0:
		vertex[tid]->setValueC(0.15f + 0.85f * sum);
		  break;
		case 1:
		vertex[tid]->setValueV(0.15f + 0.85f * sum);
		  break;
		}
      /* Write my value (divided by my out-degree) to my out-edges so neighbors
       * can read it. */
      int numOutEdge;
	 
	  switch (vertex[tid]->type) {
		case 0:
		numOutEdge = vertex[tid]->numOutEdgesC();
		  break;
		case 1:
		numOutEdge = vertex[tid]->numOutEdgesV();
				  break;
		}
	  float outValue;
	  
	  switch (vertex[tid]->type) {
		case 0:
		outValue= vertex[tid]->getValueC() / numOutEdge;
		  break;
		case 1:
		outValue= vertex[tid]->getValueV() / numOutEdge;
				  break;
		}
      for (int i = 0; i < numOutEdge; i++) {
        ChiEdge<float> *outEdge;
		
		switch (vertex[tid]->type) {
			case 0:
			outEdge = vertex[tid]->getOutEdgeC(i);
			  break;
			case 1:
			outEdge = vertex[tid]->getOutEdgeV(i);
					  break;
			}
		
		switch (outEdge->type) {
			case 0:
			outEdge->setValueC(outValue);
			  break;
			case 1:
			outEdge->setValueV(outValue);
			  break;
			}
      }
    }
  }
}

__global__ void copyBack(VirtVertex<float, float> **vertex,
                         GraphChiContext *context, float *pagerank) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < context->getNumVertices()) {
        pagerank[tid] = vertex[tid]->getValueV();
    }
}
