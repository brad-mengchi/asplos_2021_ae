#include "hip/hip_runtime.h"

#include "collision.h"
__managed__ BodyType **d_bodies;

// Helper variables for rendering and checksum computation.
__device__ int r_draw_counter = 0;
__device__ float r_Body_pos_x[kNumBodies];
__device__ float r_Body_pos_y[kNumBodies];
__device__ float r_Body_vel_x[kNumBodies];
__device__ float r_Body_vel_y[kNumBodies];
__device__ float r_Body_mass[kNumBodies];
int host_draw_counter;
float host_Body_pos_x[kNumBodies];
float host_Body_pos_y[kNumBodies];
float host_Body_vel_x[kNumBodies];
float host_Body_vel_y[kNumBodies];
float host_Body_mass[kNumBodies];
float host_Body_is_active[kNumBodies];

__device__ void Body_apply_force(IndexT id, IndexT other) {
    // Update `other`.
    if (other != id) {
        float dist;
        float F;
        CONCORDR(dist, d_bodies[other], computeDistance, (d_bodies[id]));
        CONCORDR(F, d_bodies[other], computeForce, (d_bodies[id], dist));
        CONCORD(d_bodies[other], updateForceX, (d_bodies[id], F));
        CONCORD(d_bodies[other], updateForceY, (d_bodies[id], F));
    }
}

__device__ void Body_compute_force(IndexT id) {
    CONCORD(d_bodies[id], initForce, ());

    // device_do
    for (IndexT i = 0; i < kNumBodies; ++i) {
        bool active;
        CONCORDR(active, d_bodies[i], active, ());
        if (active) {
            Body_apply_force(i, id);
        }
    }
}

__device__ void Body_update(IndexT id) {
    float idposx;
    float idposy;
    float idvelx;
    float idvely;

    CONCORD(d_bodies[id], updateVelX, ());
    CONCORD(d_bodies[id], updateVelY, ());
    CONCORD(d_bodies[id], updatePosX, ());
    CONCORD(d_bodies[id], updatePosY, ());
    CONCORDR(idposx, d_bodies[id], PosX,());
    CONCORDR(idposy, d_bodies[id], PosY,());
    CONCORDR(idvelx, d_bodies[id], VelX,());
    CONCORDR(idvely, d_bodies[id], VelY,());

    if (idposx < -1 || idposx > 1) {
        CONCORD(d_bodies[id], set_VelX, (-idvelx));
    }

    if (idposy < -1 || idposy > 1) {
        CONCORD(d_bodies[id], set_VelY, (-idvely));
    }
}

__device__ void Body_check_merge_into_this(IndexT id, IndexT other) {
    // Only merge into larger body.
    bool cond1;
    float othermass;
    float idmass;

    CONCORDR(cond1, d_bodies[other], get_incoming_merge, ());
    CONCORDR(othermass, d_bodies[other], get_mass, ());
    CONCORDR(idmass, d_bodies[id], get_mass, ());

    if (!cond1 && idmass > othermass) {
        float dist_square;
        CONCORDR(dist_square, d_bodies[id], computeDistance, (d_bodies[other]));
        dist_square *= dist_square;
        if (dist_square < kMergeThreshold * kMergeThreshold) {
            // Try to merge this one.
            // There is a race condition here: Multiple threads may try to merge
            // this body.
            CONCORD(d_bodies[id], set_merge_target, (other));
            CONCORD(d_bodies[other], set_incoming_merge, (true));
        }
    }
}

__device__ void Body_initialize_merge(IndexT id) {
    CONCORD(d_bodies[id], set_merge_target, (kNullptr));
    CONCORD(d_bodies[id], set_incoming_merge, (false));
    CONCORD(d_bodies[id], set_is_successful_merge, (false));
}

__device__ void Body_prepare_merge(IndexT id) {
    // device_do
    for (IndexT i = 0; i < kNumBodies; ++i) {
        bool active;
        CONCORDR(active, d_bodies[i], active, ());
        if (active) {
            Body_check_merge_into_this(i, id);
        }
    }
}

__device__ void Body_update_merge(IndexT id) {
    IndexT m = d_bodies[id]->get_merge_target();
    if (m != kNullptr) {
        if (d_bodies[m]->get_merge_target() == kNullptr) {
            // Perform merge.
            float mmass;
            float new_mass;
            float idvelx;
            float idvely;
            float mvelx;
            float mvely;
            float idmass;

            CONCORDR(idmass, d_bodies[id], get_mass, ());
            CONCORDR(mmass, d_bodies[m], get_mass, ());

             new_mass = idmass + mmass;
            CONCORDR(idvelx, d_bodies[id], VelX, ());
            CONCORDR(idvely, d_bodies[id], VelY, ());
            CONCORDR(mvelx, d_bodies[m], VelX, ());
            CONCORDR(mvely, d_bodies[m], VelY, ());

            float new_vel_x = (idvelx * idmass + mvelx * mmass) / new_mass;
            float new_vel_y = (idvely * idmass + mvely * mmass) / new_mass;

            CONCORD(d_bodies[m], set_mass, (new_mass));
            CONCORD(d_bodies[m], set_VelX, (new_vel_x));
            CONCORD(d_bodies[m], set_VelY, (new_vel_y));
            float idposx;
            float idposy;
            float mposx;
            float mposy;

            CONCORDR(idposx, d_bodies[id], PosX, ());
            CONCORDR(idposy, d_bodies[id], PosY, ());
            CONCORDR(mposx, d_bodies[m], PosX, ());
            CONCORDR(mposy, d_bodies[m], PosY, ());

            CONCORD(d_bodies[m], set_PosX, ((idposx + mposx) / 2));
            CONCORD(d_bodies[m], set_PosY, ((idposy + mposy) / 2));
            CONCORD(d_bodies[id], set_is_successful_merge, (true));
        }
    }
}

__device__ void Body_delete_merged(IndexT id) {
    bool cond;
    CONCORDR(cond, d_bodies[id], get_is_successful_merge, ());
    if (cond) {
        CONCORD(d_bodies[id], set_active, (false));
    }
}

__device__ void Body_add_to_draw_array(IndexT id) {
    int idx = atomicAdd(&r_draw_counter, 1);
    r_Body_pos_x[idx] = d_bodies[id]->pos_x;
    r_Body_pos_y[idx] = d_bodies[id]->pos_y;
    r_Body_vel_x[idx] = d_bodies[id]->vel_x;
    r_Body_vel_y[idx] = d_bodies[id]->vel_y;
    r_Body_mass[idx] = d_bodies[id]->mass;
}

__device__ void new_Body(IndexT id, float pos_x, float pos_y, float vel_x,
                         float vel_y, float mass) {
    d_bodies[id]->pos_x = pos_x;
    d_bodies[id]->pos_y = pos_y;
    d_bodies[id]->vel_x = vel_x;
    d_bodies[id]->vel_y = vel_y;
    d_bodies[id]->mass = mass;
    d_bodies[id]->is_active = true;
}
__global__ void kernel_initialize_bodies() {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    hiprandState rand_state;
    hiprand_init(kSeed, tid, 0, &rand_state);
    for (int id = tid; id < kNumBodies; id += blockDim.x * gridDim.x) {
        d_bodies[id] = new Body();
        assert(d_bodies[id] != NULL);

        new_Body(id,
                 /*pos_x=*/2 * hiprand_uniform(&rand_state) - 1,
                 /*pos_y=*/2 * hiprand_uniform(&rand_state) - 1,
                 /*vel_x=*/(hiprand_uniform(&rand_state) - 0.5) / 1000,
                 /*vel_y=*/(hiprand_uniform(&rand_state) - 0.5) / 1000,
                 /*mass=*/(hiprand_uniform(&rand_state) / 2 + 0.5) * kMaxMass);
    }
}

__global__ void kernel_reset_draw_counters() { r_draw_counter = 0; }

template <void (*func)(IndexT)>
__global__ void parallel_do() {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    for (int id = tid; id < kNumBodies; id += blockDim.x * gridDim.x) {
        bool active;
        CONCORDR(active, d_bodies[id], active, ());
        if (active) {
            func(id);
        }
    }
}

void transfer_data() {
    // Extract data from SoaAlloc data structure.
    kernel_reset_draw_counters<<<1, 1>>>();
    gpuErrchk(hipDeviceSynchronize());
    parallel_do<&Body_add_to_draw_array><<<kBlocks, kThreads>>>();
    gpuErrchk(hipDeviceSynchronize());

    // Copy data to host.
    hipMemcpyFromSymbol(host_Body_pos_x, HIP_SYMBOL(r_Body_pos_x),
                         sizeof(float) * kNumBodies, 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(host_Body_pos_y, HIP_SYMBOL(r_Body_pos_y),
                         sizeof(float) * kNumBodies, 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(host_Body_vel_x, HIP_SYMBOL(r_Body_vel_x),
                         sizeof(float) * kNumBodies, 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(host_Body_vel_y, HIP_SYMBOL(r_Body_vel_y),
                         sizeof(float) * kNumBodies, 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(host_Body_mass, HIP_SYMBOL(r_Body_mass),
                         sizeof(float) * kNumBodies, 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&host_draw_counter, HIP_SYMBOL(r_draw_counter), sizeof(int), 0,
                         hipMemcpyDeviceToHost);
}

int checksum() {
    transfer_data();
    int result = 0;

    for (int i = 0; i < kNumBodies; ++i) {
        int Body_checksum =
            static_cast<int>(
                (host_Body_pos_x[i] * 1000 + host_Body_pos_y[i] * 2000 +
                 host_Body_vel_x[i] * 3000 + host_Body_vel_y[i] * 4000)) %
            123456;
        result += Body_checksum;
    }

    return result;
}

int main(int /*argc*/, char ** /*argv*/) {
#ifdef OPTION_RENDER
    init_renderer();
#endif  // OPTION_RENDER

    // Allocate memory.

    hipMallocManaged(&d_bodies, sizeof(BodyType *) * kNumBodies);

    // Allocate and create Body objects.
    kernel_initialize_bodies<<<128, 128>>>();
    gpuErrchk(hipDeviceSynchronize());

#ifdef OPTION_RENDER
    // Compute max_mass.
    float max_mass = 0.0f;
    transfer_data();

    for (int i = 0; i < host_draw_counter; ++i) {
        max_mass += host_Body_mass[i];
    }
#endif  // OPTION_RENDER

    auto time_start = std::chrono::system_clock::now();

    for (int i = 0; i < kIterations; ++i) {
        printf("%i\n", i);
        parallel_do<&Body_compute_force><<<kBlocks, kThreads>>>();

        parallel_do<&Body_update><<<kBlocks, kThreads>>>();

        parallel_do<&Body_initialize_merge><<<kBlocks, kThreads>>>();

        parallel_do<&Body_prepare_merge><<<kBlocks, kThreads>>>();

        parallel_do<&Body_update_merge><<<kBlocks, kThreads>>>();

        parallel_do<&Body_delete_merged><<<kBlocks, kThreads>>>();
    }

    auto time_end = std::chrono::system_clock::now();
    auto elapsed = time_end - time_start;
    auto micros =
        std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count();

#ifndef NDEBUG
    printf("Checksum: %i\n", checksum());
    printf("#bodies: %i\n", host_draw_counter);
#endif  // NDEBUG

    printf("%lu\n", micros);

    // Free memory

#ifdef OPTION_RENDER
    close_renderer();
#endif  // OPTION_RENDER

    return 0;
}
